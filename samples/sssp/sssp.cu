#include "hip/hip_runtime.h"

#include "common.cuh"
#include "frontier.cuh"
#include "graph.cuh"
#include "graph_loader.cuh"
#include "kernel.cuh"
#include "worklist.cuh"
#include <gflags/gflags.h>
using namespace mgg;

DECLARE_int32(device);
DECLARE_string(input);
DECLARE_string(output);
DECLARE_int32(src);
DECLARE_bool(pull);
namespace sssp {

__global__ void SSSPInit(uint *label, int nnodes, vtx_t source) {
  int tid = TID_1D;
  if (tid < nnodes) {
    label[tid] = tid == source ? 0 : INFINIT;
  }
}
class job_t {
public:
  uint src;
  uint *label;
  uint itr = 0;
  vtx_t numNode;
  weight_t *adjwgt = nullptr;
  void operator()(vtx_t _numNode, uint _src, weight_t *_adjwgt) {
    numNode = _numNode;
    src = _src;
    adjwgt = _adjwgt;
    init();
  }
  void init() {
    H_ERR(hipMallocManaged(&label, numNode * sizeof(uint)));
    SSSPInit<<<numNode / BLOCK_SIZE + 1, BLOCK_SIZE>>>(label, numNode, src);
  }
  void prepare(){}
    void clean() {
  // __host__ __device__ ~job_t() {
#if !defined(__CUDA_ARCH__)
    if (!gflags::GetCommandLineFlagInfoOrDie("output").is_default)
      print::SaveResults(FLAGS_output, label, numNode);
#endif
  }
};

struct updater {
  __forceinline__ __device__ bool operator()(vtx_t src, vtx_t dst,
                                             vtx_t edge_id, job_t job) {
    if (job.label[dst] > job.label[src] + job.adjwgt[edge_id]) {
      job.label[dst] = job.label[src] + job.adjwgt[edge_id];
      return true;
    }
    return false;
  }
};
struct generator {
  __forceinline__ __device__ void operator()(bool updated,
                                             worklist::Worklist wl, vtx_t dst) {
    if (updated)
      wl.append(dst);
  }
  __forceinline__ __device__ void operator()(bool updated, char *flag,
                                             vtx_t dst) {
    if (updated)
      flag[dst] = true;
  }
  __forceinline__ __device__ void operator()(bool updated, char *flag,
                                             vtx_t dst, char *finished) {
    if (updated) {
      flag[dst] = true;
      *finished = false;
    }
  }
};
struct pull_selector {
  __forceinline__ __device__ bool operator()(vtx_t id, job_t job) {
    // if (job.label[id] == INFINIT) {
    return true;
    // }
    // return false;
  }
};

} // namespace sssp
bool SSSP_multi_gpu() {}
bool SSSP_pull_single_gpu() {
  hipSetDevice(FLAGS_device);
  H_ERR(hipDeviceReset());
  graph_t<CSR> G_csr;
  graph_loader loader;
  loader.Load(G_csr, false);
  graph_t<CSC> G;
  G.CSR2CSC(G_csr);
  LOG("SSSP pull single\n");
  hipStream_t stream;
  hipStreamCreate(&stream);
  sssp::job_t job;
  job(G.numNode, FLAGS_src, G.adjwgt);
  frontier::Frontier<BITMAP> F; // BDF  BDF_AUTO BITMAP
  F.Init(G.numNode, FLAGS_src, FLAGS_device, 1.0, false);
  G.Set_Mem_Policy(&stream); // stream
  hipDeviceSynchronize();
  Timer t;
  t.Start();
  kernel_pull<sssp::updater, sssp::generator, sssp::pull_selector, sssp::job_t>
      K;
  while (!F.finish()) {
    // cout << "itr " << job.itr << " wl_sz " << F.wl_sz << endl;
    K(G, F, job);
    hipDeviceSynchronize();
    // H_ERR(hipStreamSynchronize(stream));
    F.Next();
    job.itr++;
  }
  cout << "itr " << job.itr << " in " << t.Finish() << endl;
  return 0;
}
bool SSSP_single_gpu() {
  if (FLAGS_pull) {
    return SSSP_pull_single_gpu();
  }
  hipSetDevice(FLAGS_device);
  H_ERR(hipDeviceReset());
  graph_t<CSR> G(true);
  graph_loader loader;
  loader.Load(G, true);
  // LOG("make g1 chunks\n");
  // G.make_chunks(4);
  // for (size_t i = 0; i < 4; i++) {
  //   cout << "G " << i << G.chunks[i] << endl;
  // }
  // graph_t<CSC> G2;
  // G2.CSR2CSC(G);
  // G2.make_chunks(4);
  // for (size_t i = 0; i < 4; i++) {
  //   cout << "G2 " << i << G2.chunks[i] << endl;
  // }

  LOG("SSSP single\n");
  hipStream_t stream;
  hipStreamCreate(&stream);
  // G.Init(false);
  sssp::job_t job;
  job(G.numNode, FLAGS_src, G.adjwgt);
  frontier::Frontier<BDF_AUTO> F; // BDF  BDF_AUTO BITMAP
  F.Init(G.numNode, FLAGS_src, FLAGS_device, 1.0, false);
  G.Set_Mem_Policy(&stream); // stream
  hipDeviceSynchronize();
  Timer t;
  t.Start();
  kernel<graph_t<CSR>, frontier::Frontier<BDF_AUTO>, sssp::updater,
         sssp::generator, sssp::job_t>
      K;
  while (!F.finish()) {
    // cout << "itr " << job.itr << " wl_sz " << F.wl_sz << endl;
    K(G, F, job);
    hipDeviceSynchronize();
    // H_ERR(hipStreamSynchronize(stream));
    F.Next();
    job.itr++;
  }
  cout << "itr " << job.itr << " in " << t.Finish() << endl;
  job.clean();
  return 0;
}
