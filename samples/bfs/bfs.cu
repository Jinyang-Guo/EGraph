#include "hip/hip_runtime.h"

#include "common.cuh"
#include "frontier.cuh"
#include "graph.cuh"
#include "graph_loader.cuh"
#include "kernel.cuh"
#include "worklist.cuh"
#include <gflags/gflags.h>
using namespace mgg;

DECLARE_int32(device);
DECLARE_string(input);
DECLARE_string(output);
DECLARE_int32(src);
DECLARE_bool(pull);
namespace bfs {

__global__ void BFSInit(uint *label, int nnodes, vtx_t source) {
  int tid = TID_1D;
  if (tid < nnodes) {
    label[tid] = tid == source ? 0 : INFINIT;
  }
}
// template<typename graph_t>
class job_t {
public:
  uint src;
  uint *label;
  uint itr = 0;
  vtx_t numNode;
  weight_t *adjwgt = nullptr;
  void operator()(vtx_t _numNode, uint _src) {
    numNode = _numNode;
    src = _src;
    init();
  }
  void init() {
    H_ERR(hipMallocManaged(&label, numNode * sizeof(uint)));
    BFSInit<<<numNode / BLOCK_SIZE + 1, BLOCK_SIZE>>>(label, numNode, src);
  }
  void prepare() {}
  void clean() {
// __host__ __device__ ~job_t() {
#if !defined(__CUDA_ARCH__)
    if (!gflags::GetCommandLineFlagInfoOrDie("output").is_default)
      print::SaveResults(FLAGS_output, label, numNode);
#endif
  }
};

struct updater {
  __forceinline__ __device__ bool operator()(vtx_t src, vtx_t dst,
                                             vtx_t edge_id, job_t job) {
    if (job.label[dst] > job.itr + 1) {
      job.label[dst] = job.itr + 1;
      return true;
    }
    return false;
  }
};
struct generator {
  __forceinline__ __device__ void operator()(bool updated,
                                             worklist::Worklist wl, vtx_t dst) {
    if (updated)
      wl.append(dst);
  }
  __forceinline__ __device__ void operator()(bool updated, char *flag,
                                             vtx_t dst) {
    if (updated)
      flag[dst] = true;
  }
  __forceinline__ __device__ void operator()(bool updated, char *flag,
                                             vtx_t dst, char *finished) {
    if (updated) {
      flag[dst] = true;
      *finished = false;
    }
  }
};
struct pull_selector {
  __forceinline__ __device__ bool operator()(vtx_t id, job_t job) {
    if (job.label[id] == INFINIT) {
      return true;
    }
    return false;
  }
};
} // namespace bfs
bool BFS_multi_gpu() {
  graph_t<CSR> G_csr;
  graph_loader loader;
  loader.Load(G_csr, false);
  graph_t<CSC> G;
  G.CSR2CSC(G_csr);
  G.make_chunks(4);
  // for (size_t i = 0; i < 4; i++) {
  //   cout << "G " << i << G.chunks[i] << endl;
  // }
  hipStream_t stream;
  hipStreamCreate(&stream);
  LOG("distributing\n");
  G.distribute_chunks(&stream);
}
bool BFS_pull_single_gpu() {
  hipSetDevice(FLAGS_device);
  H_ERR(hipDeviceReset());
  graph_t<CSR> G_csr;
  graph_loader loader;
  loader.Load(G_csr, false);
  graph_t<CSC> G;
  G.CSR2CSC(G_csr);
  LOG("BFS pull single\n");
  hipStream_t stream;
  hipStreamCreate(&stream);
  bfs::job_t job;
  job(G.numNode, FLAGS_src);
  frontier::Frontier<BITMAP> F; // BDF  BDF_AUTO BITMAP
  F.Init(G.numNode, FLAGS_src, FLAGS_device, 1.0, false);
  G.Set_Mem_Policy(&stream); // stream
  hipDeviceSynchronize();
  Timer t;
  t.Start();
  kernel_pull<bfs::updater, bfs::generator, bfs::pull_selector, bfs::job_t> K;
  while (!F.finish()) {
    // cout << "itr " << job.itr << " wl_sz " << F.wl_sz << endl;
    K(G, F, job);
    hipDeviceSynchronize();
    // H_ERR(hipStreamSynchronize(stream));
    F.Next();
    job.itr++;
  }
  cout << "itr " << job.itr << " in " << t.Finish() << endl;
  return 0;
}
bool BFS_single_gpu() {
  if (FLAGS_pull) {
    return BFS_pull_single_gpu();
  }
  hipSetDevice(FLAGS_device);
  // H_ERR(hipDeviceReset());
  graph_t<CSR> G;
  graph_loader loader;
  loader.Load(G, false);
  LOG("BFS single\n");
  hipStream_t stream;
  hipStreamCreate(&stream);
  // G.Init(false);
  bfs::job_t job;
  job(G.numNode, FLAGS_src);
  frontier::Frontier<BDF> F; // BDF  BDF_AUTO BITMAP WL
  F.Init(G.numNode, FLAGS_src, FLAGS_device, 1.0, false);
  Timer totalT;
  totalT.Start();
  G.Set_Mem_Policy(&stream); // stream
  hipStreamSynchronize(stream);
  kernel<graph_t<CSR>, frontier::Frontier<BDF>, bfs::updater,
         bfs::generator, bfs::job_t>
      K;
  Timer t;
  t.Start();

  while (!F.finish()) {
    // cout << "itr " << job.itr << " wl_sz " << F.wl_sz << endl;
    K(G, F, job);
    // hipDeviceSynchronize();
    // H_ERR(hipStreamSynchronize(stream));
    F.Next();
    job.itr++;
  }
  cout << "itr " << job.itr << " in " << t.Finish() <<" totoal "<< totalT.Finish() << endl;
  job.clean();
  return 0;
}
